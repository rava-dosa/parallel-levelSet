
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string.h>


__device__ void reInitialization(double *U);
//kernel
__global__ void finiteD_gpu(double *U, double dt, double vn, int totalStep)
{		
	reInitialization(U);
}


__device__ void reInitialization(double *U)
{
        double d_dh=0.2;
	int x=threadIdx.x;
	int y=blockIdx.x;
	int z=blockIdx.y;
	
	int index=z*blockDim.x*gridDim.x+y*blockDim.x+x;
	
	double coords[3];
	coords[0]=x*d_dh;
	coords[1]=y*d_dh;
	coords[2]=z*d_dh;
	
	double tem_phi;
	if(coords[2]>5)tem_phi=std::sqrt((coords[0]-3)*(coords[0]-3)+(coords[1]-5)*(coords[1]-5)+(coords[2]-7)*(coords[2]-7))-1;
	else tem_phi=std::sqrt((coords[0]-7)*(coords[0]-7)+(coords[1]-5)*(coords[1]-5)+(coords[2]-3)*(coords[2]-3))-1;
U[index]=tem_phi;
	//U[index]=fmin(tem_phi1,tem_phi2);
	
}

__global__ void update_phi(double *U,double dt, double vn, int step)
{
	double d_dh=0.2;

	int my_x=threadIdx.x;
	int my_y=blockIdx.x;
	int my_z=blockIdx.y;
	
	int nei_x,nei_y,nei_z;
	int nei_index;
	
	int dof=blockDim.x*gridDim.x*gridDim.y;
	
	//start from time step i for U
	int index=dof*step+my_z*blockDim.x*gridDim.x+my_y*blockDim.x+my_x;
	double my_phi=U[index-dof];
	
	//
	double dmx, dpx,dmy,dpy,dmz,dpz,fluxx,fluxy,fluxz;

	nei_x=my_x-1;
	nei_y=my_y;
	nei_z=my_z;
	//
	nei_index=dof*(step-1)+nei_z*blockDim.x*gridDim.x+nei_y*blockDim.x+nei_x;
	if(nei_x<0) dmx=0;
	else dmx=(my_phi-U[nei_index])/d_dh;
	
	nei_x=my_x+1;
	nei_y=my_y;
	nei_z=my_z;
	//
	nei_index=dof*(step-1)+nei_z*blockDim.x*gridDim.x+nei_y*blockDim.x+nei_x;
	if(nei_x>blockDim.x-1) dpx=0;
	else dpx=(U[nei_index]-my_phi)/d_dh;
	
	fluxx=fmax(dmx,0.0)*fmax(dmx,0.0)+fmin(dpx,0.0)*fmin(dpx,0.0);
	//--------------------------
	
	nei_x=my_x;
	nei_y=my_y-1;
	nei_z=my_z;
	//
	nei_index=dof*(step-1)+nei_z*blockDim.x*gridDim.x+nei_y*blockDim.x+nei_x;
	if(nei_y<0) dmy=0;
	else dmy=(my_phi-U[nei_index])/d_dh;
	
	nei_x=my_x;
	nei_y=my_y+1;
	nei_z=my_z;
	//
	nei_index=dof*(step-1)+nei_z*blockDim.x*gridDim.x+nei_y*blockDim.x+nei_x;
	if(nei_y>gridDim.x-1) dpy=0;
	else dpy=(U[nei_index]-my_phi)/d_dh;
	
	fluxy=fmax(dmy,0.0)*fmax(dmy,0.0)+fmin(dpy,0.0)*fmin(dpy,0.0);
	//--------------------------
	
	nei_x=my_x;
	nei_y=my_y;
	nei_z=my_z-1;
	//
	nei_index=dof*(step-1)+nei_z*blockDim.x*gridDim.x+nei_y*blockDim.x+nei_x;
	if(nei_z<0) dmz=0;
	else dmz=(my_phi-U[nei_index])/d_dh;
	
	nei_x=my_x;
	nei_y=my_y;
	nei_z=my_z+1;
	//
	nei_index=dof*(step-1)+nei_z*blockDim.x*gridDim.x+nei_y*blockDim.x+nei_x;
	if(nei_z>gridDim.y-1) dpz=0;
	else dpz=(U[nei_index]-my_phi)/d_dh;
	
	fluxz=fmax(dmz,0.0)*fmax(dmz,0.0)+fmin(dpz,0.0)*fmin(dpz,0.0);
	
	
	U[index]=my_phi-vn*sqrt(fluxx+fluxy+fluxz)*dt;
	
}
