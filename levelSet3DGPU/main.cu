#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string.h>
#include <ctime>
#include <vector>
#include "include/finiteDmesh.h"
#include "include/solutionClass.h"
#include "include/DataOut.h"

//Constant
#define dim 3


using namespace std;



//#include "levelSet_GPU_kernel.h"

#define THREADS_PER_BLOCK 512

__global__ void finiteD_gpu(double *U, double dt, double vn, int totalStep);
__global__ void update_phi(double *U, double dt, double vn, int step);
int main(int argc, char *argv[]){
  std::clock_t start=std::clock();
	int num_point=51;
	//define output for 101*101*101 dof with 20 timesteps
	std::vector<double> point0(dim), point1(dim);
	std::vector<int> ControlPointperDim;
	ControlPointperDim.resize(dim);
	for(unsigned int i=0;i<dim;i++) {
		point0[i]=0.0; point1[i]=10.0;
		ControlPointperDim[i]=num_point;
	}

	finiteDmesh<dim> mesh;
	mesh.generateControlPoint(rectangular, point0, point1, ControlPointperDim);
	//solution class
	solutionClass<dim> phi(mesh, NODAL, SCALAR, std::string("phi"));
	//outputData class
	DataOut<dim> outputData;
	outputData.generateOutputGrid(&mesh);
	
	
	
	std::cout<<"aa"<<std::endl;	
	int totalTimestep=10;
	int dof=num_point*num_point*num_point;
	double *U;
	double *d_U;
	
	//need one more to store initial value
	int size=dof*(totalTimestep+1)*sizeof(double);	
	hipMalloc(&d_U, size);
	U=(double *)malloc(size);
	
	
	//setting for easy index
	dim3 threadsPerBlock(num_point,1,1);
	dim3 numBlcoks(num_point,num_point);
	
std::clock_t GPU_time=std::clock();
	finiteD_gpu<<<numBlcoks, threadsPerBlock>>>(d_U,0.05 ,1,totalTimestep);

        for(int i=1;i<=totalTimestep;i++){
                update_phi<<<numBlcoks, threadsPerBlock>>>(d_U,0.05, 1, i);
       
        }
hipMemcpy(U, d_U, size, hipMemcpyDeviceToHost);
	printf ("\nTime taken:%10.2e sec\n", (std::clock()-GPU_time)/((double)CLOCKS_PER_SEC));
	//output
	for(unsigned int s=0;s<=totalTimestep;s++){
  	for (unsigned int i=0; i<dof; i++){
			phi.values[i]=U[i+dof*s];
  	}
		std::vector<solutionClass<dim>* > outputVariables;
		outputVariables.push_back(&phi);
  	char fileName[200];
		std::sprintf (fileName, "../output/output");
  	//std::sprintf (fileName, "output0");
		outputData.writeMesh(fileName,  s,outputVariables);
	}
	
	
	
	free(U);
	hipFree(d_U);
	
  //Stats
  printf ("\nTime taken:%10.2e sec\n", (std::clock()-start)/((double)CLOCKS_PER_SEC));
}

